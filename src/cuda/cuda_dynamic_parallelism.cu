#include "hip/hip_runtime.h"
#include <cuda/std/complex>
#include <cuda/functional>
#include <complex>
#include <iostream>
#include <vector>
#include <string>
#include <assert.h>
#include <hip/hip_complex.h>

//////////////////////////////////////////////////////////////
// CUDA ERROR HANDLING
//////////////////////////////////////////////////////////////
void checkAsync(hipError_t err, std::string position = "") {
    if (err != hipSuccess) {
        std::cerr << "Cuda Runtime Error at " << position << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

void checkSync(std::string position = "") {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Cuda Runtime Error at " << position << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

//////////////////////////////////////////////////////////////
// NECESSARY FUNCTIONS FOR DFT
//////////////////////////////////////////////////////////////
__device__ constexpr cuda::std::complex<double> operator""_i(long double d) {
    return cuda::std::complex<double>{0.0, static_cast<double>(d)};
}

__device__ constexpr cuda::std::complex<double> pi() {
    return atan(1.0) * 4;
}

void printMatrix(double* matrix, int* matSize) {
    int size = *matSize;
    std::cout << size << std::endl;

    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            std::cout << matrix[i * size + j] << " ";
        }
        std::cout << std::endl;
    }
}

void printResult(cuda::std::complex<double>* result, int size) {
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            // std::cout << "(" << result[i * size + j].real() << ", " << result[i * size + j].imag() << ")" << " ";
            printf("(%.6lf, %.6lf) ", result[i * size + j].real(), result[i * size + j].imag());
        }
        std::cout << std::endl;
    }
}

__device__ cuda::std::complex<double> dftElement(double* source, int size, int k, int l) {
    cuda::std::complex<double> element(0.0, 0.0);

    for (int m = 0; m < size; m++) {
        for (int n = 0; n < size; n++) {
            cuda::std::complex<double> sample = (k * m / (double)size) + (l * n / (double)size);
            cuda::std::complex<double> exponent = exp(-2.0_i * pi() * sample);
            element += source[m * size + n] * exponent;
        }
    }

    return element / (cuda::std::complex<double>)(size * size);
}

//////////////////////////////////////////////////////////////
// KERNEL FUNCTION
//////////////////////////////////////////////////////////////
__global__ void device_hello_world() {
    printf("Hello world from x.%d y.%d z.%d!\n", threadIdx.x, threadIdx.y, threadIdx.z);
}

__global__ void dft_sub_kernel(double* source, int size, int gridRow, int gridCol, hipDoubleComplex* sum, cuda::std::complex<double>* result) {
    int row = gridRow * 32 + blockIdx.y * blockDim.y + threadIdx.y;
    int col = gridCol * 32 + blockIdx.x * blockDim.x + threadIdx.x;

    if ((row >= size) || (col >= size)) {
        return;
    }

    result[row * size + col] = dftElement(source, size, row, col);

    atomicAdd(&(sum->x), result[row * size + col].real());
    atomicAdd(&(sum->y), result[row * size + col].imag());
}

__global__ void dft_kernel(double* source, int size, hipDoubleComplex* sum, cuda::std::complex<double>* result) {
    dim3 threads_per_block(32, 32, 1);
    dim3 blocks_per_grid(1, 1, 1);

    hipStream_t s;
    hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
    dft_sub_kernel << <blocks_per_grid, threads_per_block, 0, s >> > (source, size, threadIdx.x, threadIdx.y, sum, result);
    hipStreamDestroy(s);
}

//////////////////////////////////////////////////////////////
// MAIN FUNCTION. LET'S GO!!!
//////////////////////////////////////////////////////////////
int main(void) {
    int size;
    double* source_host;

    // Read Matrix
    std::cin >> size;
    source_host = (double*)malloc(size * size * sizeof(double));
    for (int i = 0; i < size * size; i++) {
        std::cin >> source_host[i];
    }

    // Print Matrix
    // printMatrix(source_host, &size);

    double* source_gpu;
    hipDoubleComplex sum_host = make_hipDoubleComplex(0.0, 0.0);
    hipDoubleComplex* sum_gpu;

    cuda::std::complex<double>* result_gpu;
    cuda::std::complex<double>* result_host = (cuda::std::complex<double>*)malloc(size * size * sizeof(cuda::std::complex<double>));

    checkAsync(hipMalloc(&source_gpu, sizeof(double) * size * size), "Malloc source to GPU");
    checkAsync(hipMemcpy(source_gpu, source_host, sizeof(double) * size * size, hipMemcpyHostToDevice), "Memcpy source to GPU");

    checkAsync(hipMalloc(&sum_gpu, sizeof(hipDoubleComplex)), "Malloc sum to GPU");
    checkAsync(hipMemcpy(sum_gpu, &sum_host, sizeof(hipDoubleComplex), hipMemcpyHostToDevice), "Memcpy sum to GPU");

    checkAsync(hipMalloc(&result_gpu, sizeof(cuda::std::complex<double>) * size * size), "Malloc result to GPU");
    checkAsync(hipMemcpy(result_gpu, result_host, sizeof(double) * size * size, hipMemcpyHostToDevice), "Memcpy result to GPU");

    int block_count = std::ceil((double)(size) / (double)(32));
    dim3 threads_per_block(block_count, block_count, 1);
    dim3 blocks_per_grid(1, 1, 1);

    dft_kernel << <blocks_per_grid, threads_per_block >> > (source_gpu, size, sum_gpu, result_gpu);

    hipDeviceSynchronize();

    checkAsync(hipMemcpy(&sum_host, sum_gpu, sizeof(hipDoubleComplex), hipMemcpyDeviceToHost), "Memcpy sum to HOST");
    checkAsync(hipMemcpy(result_host, result_gpu, sizeof(cuda::std::complex<double>) * size * size, hipMemcpyDeviceToHost), "Memcpy result to HOST");

    printResult(result_host, size);
    sum_host.x /= size;

    // std::cout << "Average : (" << sum_host.real() << "," << sum_host.imag() << ")" << std::endl;
    printf("Average: (%.6lf, %.6lf)\n", hipCreal(sum_host), hipCimag(sum_host));

    checkAsync(hipFree(source_gpu), "Free source");
    checkAsync(hipFree(result_gpu), "Free result");
    checkAsync(hipFree(sum_gpu), "Free sum");

    delete[] source_host;
    delete[] result_host;
    return 0;
}